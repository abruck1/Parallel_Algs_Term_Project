#include "cuda_fft.h"
#include <iostream>
#include <stdlib.h>
using std::cout;
using std::endl;


void fft_cuda(thCdouble* h_A, int n, direction dir)
{
	// define device data
	thCdouble *d_A;
	hipMalloc((thCdouble**) &d_A, n*sizeof(thCdouble));
	thCdouble *d_B;
	hipMalloc((thCdouble**) &d_B, n*sizeof(thCdouble));
	hipMemcpy(d_B, h_A, n*sizeof(thCdouble), hipMemcpyHostToDevice);

	// invoke kernel
    int threadsPerBlock = MIN(n,MAX_THREADS);
    int nBlocks = (n-1)/threadsPerBlock + 1;

	// block level kernel call
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 1<<24);
	bit_reverse_kernel<<<nBlocks,threadsPerBlock>>>(d_A, d_B, n);
    hipDeviceSynchronize();
    fft_kernel_shared<<<nBlocks,threadsPerBlock,threadsPerBlock*sizeof(thCdouble)>>>(d_A, n, dir);
    hipDeviceSynchronize();
	cout << endl;

	// continue FFT in global memory

	if(nBlocks>1)
		for(int m=2*MAX_THREADS; m<=n; m<<=1)
		{
			cout << "Going again with m = " << m << endl;
			fft_kernel_finish<<<nBlocks, threadsPerBlock>>>(d_A, m, dir);
			hipDeviceSynchronize();
		}


	// copy back to host
    hipMemcpy(h_A, d_A, n*sizeof(thCdouble), hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
}



void fft_cuda(thCdouble* h_A, int n)
{
	fft_cuda(h_A, n, FORWARD);
}



void ifft_cuda(thCdouble* h_A, int n)
{
	fft_cuda(h_A, n, REVERSE);
}


int main(int argc, char** argv)
{
	// define host data
	int ex = atoi(argv[1]);
	int n = 1<<ex;
	thCdouble *h_A = new thCdouble[n];
	for(int i=0; i<n; i++)
		h_A[i] = i+1;	

	// call fft
	fft_cuda(h_A, n);

	// print result
//	for(int i=0; i<n; i++)
//		cout << h_A[i] << endl;

	// memory cleanup
	delete[] h_A;
}
