#include "fft_cuda.h"
#include "fft_cuda_kernels.h"
#include <stdio.h>


void fft_cuda_transform(thCdouble* h_A, int n, direction dir)
{
	// define device data
	thCdouble *d_A;
	hipMalloc((thCdouble**) &d_A, n*sizeof(thCdouble));
	thCdouble *d_B;
	hipMalloc((thCdouble**) &d_B, n*sizeof(thCdouble));
	hipMemcpy(d_B, h_A, n*sizeof(thCdouble), hipMemcpyHostToDevice);

	// invoke kernel
    int threadsPerBlock = MIN(n,MAX_THREADS);
    int nBlocks = (n-1)/threadsPerBlock + 1;

	// block level kernel call
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 1<<24);
	bit_reverse_kernel<<<nBlocks,threadsPerBlock>>>(d_A, d_B, n);
    hipDeviceSynchronize();
    fft_kernel_shared<<<nBlocks,threadsPerBlock,threadsPerBlock*sizeof(thCdouble)>>>(d_A, n, dir);
    hipDeviceSynchronize();

	// continue FFT in global memory

	if(nBlocks>1)
		for(int m=2*MAX_THREADS; m<=n; m<<=1)
		{
			fft_kernel_finish<<<nBlocks, threadsPerBlock>>>(d_A, m, dir);
			hipDeviceSynchronize();
		}


	// copy back to host
    hipMemcpy(h_A, d_A, n*sizeof(thCdouble), hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
}



void fft_cuda(thCdouble* h_A, int n)
{
	fft_cuda_transform(h_A, n, FORWARD);
}



void ifft_cuda(thCdouble* h_A, int n)
{
	fft_cuda_transform(h_A, n, REVERSE);
}




